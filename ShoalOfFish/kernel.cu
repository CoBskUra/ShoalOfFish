#include "hip/hip_runtime.h"
﻿// ilość ryb w kompilatorze ustalana 
// pomysł z tablicą na karzdy sektor jest słaby
// trzeba jakość segregować rybki w zależności od grupy
#include "hip/hip_runtime.h"
#include ""

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <cmath>
#define width 1280   //screen width
#define height 700   //screen height
#define fishNumber 1000
#define maxThreds 1024
#define maxBlocks 100
#define M_PI 3.14159265358979323846
#define MaxSpeed 10

constexpr float CohesionScale = 0.01f;
constexpr float AlignmentScale = 0.1f;
constexpr float SeparationScale = 0.1f;


struct Shoal {
    float* position_x;
    float* position_y;
    float* velocity_x;
    float* velocity_y;
    int h = 20;
    int w = 5;
    int minDistance = 20;
    int viewRange = 100;
};

struct Point {
    double x;
    double y;
};

int x = 0;
float t = 0.0f;
float* device;   //pointer to memory on the device (GPU VRAM)
GLuint buffer;   //buffer
Shoal shoal;
float background;
float fishColor;

__device__ int sign(Point p1, Point p2, Point p3)
{
    return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
}

__device__ Point MakePoint(float x, float y)
{
    Point p;
    p.x = x;
    p.y = y;
    return p;
}

__device__ double Distance(Point p1, Point p2)
{
    return pow(p1.x - p2.x, (double)2) + pow(p1.y - p2.y, (double)2);
}

__device__ float Direction(Point vel)
{
    float direction = asin(vel.y / sqrt(Distance(MakePoint(0, 0), vel)));
    
    if (vel.x < 0)
    {
        if (vel.y >= 0)
        {
            float alfa = M_PI / 2 - direction;
            direction += 2 * alfa;
        }
        else
        {
            float alfa = M_PI / 2 + direction;
            direction -= 2 * alfa;
        }
    }
    return direction;
}

__device__ void FishToCordinates(Point fishPosition, float direction, int h, int w, Point* p1, Point* p2, Point* p3)
{
    float hX = cos(direction) * h / 2;
    float hY = sin(direction) * h / 2;

    p1->x = fishPosition.x + hX;
    p1->y = fishPosition.y + hY;

    float fishWingX = -sin(direction) * w;
    float fishWingY = cos(direction) * w;

    p2->x = fishPosition.x - hX + fishWingX;
    p2->y = fishPosition.y - hY + fishWingY;

    p3->x = fishPosition.x - hX - fishWingX;
    p3->y = fishPosition.y - hY - fishWingY;
}
__global__ void CalculateShoal(Shoal shoal, float* output)
{
    unsigned int fishId = blockIdx.x * blockDim.x + threadIdx.x;
    if (fishId > fishNumber)
        return;

    float viewRange = shoal.viewRange * shoal.viewRange;
    float minDistance = shoal.minDistance * shoal.minDistance;
    
    
        Point bois = MakePoint(shoal.position_x[fishId], shoal.position_y[fishId]);
        Point centerOfMass = MakePoint(0,0);
        Point separation = MakePoint(0, 0);
        Point avrVelocity = MakePoint(0, 0);
        int neighboursCountCenterOfMass = 0;
        int neighboursCountVelocity = 0;
        for (int neighbourFishId = 0; neighbourFishId < fishNumber; neighbourFishId++)
        {
            if (neighbourFishId == fishId)
                continue;

            Point boisFriend = MakePoint(shoal.position_x[neighbourFishId], shoal.position_y[neighbourFishId]);
            double distance = Distance(bois, boisFriend);
            if (viewRange > distance)
            {
                neighboursCountCenterOfMass++;
                centerOfMass.x += boisFriend.x;
                centerOfMass.y += boisFriend.y;

            }

            if (viewRange > distance)
            {
                neighboursCountVelocity++;
                avrVelocity.x += shoal.velocity_x[neighbourFishId];
                avrVelocity.y += shoal.velocity_x[neighbourFishId];
            }
            if (minDistance > distance)
            {
                separation.x -= boisFriend.x - bois.x;
                separation.y -= boisFriend.y - bois.y;
            }
        }
        Point newPosition = MakePoint(shoal.velocity_x[fishId], shoal.velocity_y[fishId]);
        if (neighboursCountCenterOfMass > 0)
        {
            centerOfMass.x /= neighboursCountCenterOfMass;
            centerOfMass.y /= neighboursCountCenterOfMass;

             newPosition.x += (centerOfMass.x - bois.x) * CohesionScale;
             newPosition.y += (centerOfMass.y - bois.y) * CohesionScale;
        }

        separation.x *= SeparationScale;
        separation.y *= SeparationScale;

        newPosition.x += separation.x;
        newPosition.y += separation.y;

        if (neighboursCountVelocity != 0) {
            avrVelocity.x /= neighboursCountVelocity;
            avrVelocity.y /= neighboursCountVelocity;
            newPosition.x += avrVelocity.x * AlignmentScale;
            newPosition.y += avrVelocity.y * AlignmentScale;
        }

        if (Distance(MakePoint(0, 0), newPosition) > MaxSpeed * MaxSpeed)
        {
            double calculateSpeed = sqrt( Distance(MakePoint(0, 0), newPosition));
            newPosition.x *= MaxSpeed / calculateSpeed;
            newPosition.y *= MaxSpeed / calculateSpeed;
        }
    

    shoal.position_x[fishId] += newPosition.x;
    shoal.position_y[fishId] += newPosition.y;
    shoal.velocity_x[fishId] = newPosition.x;
    shoal.velocity_y[fishId] = newPosition.y;

    if (shoal.position_x[fishId] < 0)
        shoal.position_x[fishId] = width;
    else if (shoal.position_x[fishId] > width)
        shoal.position_x[fishId] = 0;

    if (shoal.position_y[fishId] < 0)
        shoal.position_y[fishId] = height;
    else if (shoal.position_y[fishId] > height)
        shoal.position_y[fishId] = 0;

    Point p1, p2, p3;
    FishToCordinates(bois, Direction(newPosition), shoal.h, shoal.w, &p1, &p2, &p3);
    int start = fishId * 6;
    output[start] = p1.x;
    output[start + 1] = p1.y;
    output[start + 2] = p2.x;
    output[start + 3] = p2.y;
    output[start + 4] = p3.x;
    output[start + 5] = p3.y;
}



__global__ void InitStartPosition(Shoal shoal, int fisheGrideWidth, int fishGrideHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    int distance = shoal.minDistance;
    int startPoint_x = (width - distance*fisheGrideWidth) / 2;
    int startPoimt_y = (height - distance*fishGrideHeight) / 2;
    int fishGridY = x / fisheGrideWidth;
    int fishGridX = x - fishGridY * fisheGrideWidth;
    shoal.velocity_x[x] = MaxSpeed * cos(x * 2* M_PI / fishNumber);
    shoal.velocity_y[x] = MaxSpeed * sin(x * 2 * M_PI / fishNumber);
    shoal.position_x[x] = startPoint_x + fishGridX * distance;
    shoal.position_y[x] = startPoimt_y + fishGridY * distance;
}

// Global variables to track the position of the dot


void time(int x)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(10, time, 0);
        t += 0.0166f;
    }
}

void CalculateNeededThreads(int* threads, int* blocks)
{
    if (fishNumber < maxThreds)
    {
        *threads = fishNumber;
        *blocks = 1;
    }
    else
    {
        *threads = maxThreds;
        *blocks = ceil(fishNumber / maxThreds);
        if (*blocks > maxBlocks)
            exit(-1);
    }
}

void LunchCuda()
{
    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks);
    CalculateShoal << <blocks, threads >> > (shoal, device);
    hipDeviceSynchronize();

}
// Display callback function
void display() {
    // Clear the window
    cudaGLMapBufferObject((void**)&device, buffer);   //maps the buffer object into the address space of CUDA
    glClear(GL_COLOR_BUFFER_BIT);

    LunchCuda();
    hipDeviceSynchronize();

    cudaGLUnmapBufferObject(buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    /*glVertexPointer(2, GL_FLOAT, 12, 0);
    glColorPointer(4, GL_UNSIGNED_BYTE, 12, (GLvoid*)8);
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glDrawArrays(GL_POINTS, 0, width * height);
    glDisableClientState(GL_VERTEX_ARRAY);*/
    glDrawArrays(GL_TRIANGLES, 0, 3*fishNumber);
    glutSwapBuffers();
    
    x++;
    if (x > width)
        x = 0;
}

void InitCuda()
{
    hipMalloc(&device, fishNumber * 6 * sizeof(float));   //allocate memory on the GPU VRAM
    hipMalloc(&shoal.position_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.position_y, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_y, fishNumber * sizeof(float));
    int fisheGrideWidth = ceil(sqrt((fishNumber * width) / height));
    int fishGrideHeight = ceil(fisheGrideWidth * height / width);

    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks);
    InitStartPosition << <blocks, threads >> > (shoal, fisheGrideWidth, fishGrideHeight);
}

void Init()
{
    glClearColor(0.0, 0.0, 0.0, 0.0);
    glMatrixMode(GL_PROJECTION);
    gluOrtho2D(0.0, width, 0.0, height);
    glutDisplayFunc(display);
    //glutReshapeFunc(Reshape);
    time(0);
    glewInit();
    glGenBuffers(1, &buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    unsigned int size = fishNumber * 6 * sizeof(float); // ilość wektorów
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 2, 0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    InitCuda();
    cudaGLRegisterBufferObject(buffer);   //register the buffer object for access by CUDA
}

int main(int argc, char** argv) {
    // Initialize GLUT
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);   //display mode
    glutInitWindowSize(width, height);
    glutCreateWindow("Moving Dot"); // Create the window
    Init();
    // Enter the main loop
    glutMainLoop();
    hipFree(device);
    return 0;
}