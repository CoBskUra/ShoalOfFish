#include "hip/hip_runtime.h"
﻿// ilość ryb w kompilatorze ustalana 
// pomysł z tablicą na karzdy sektor jest słaby
// trzeba jakość segregować rybki w zależności od grupy
#include "hip/hip_runtime.h"
#include ""

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <cmath>
#include <string>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#define width 1280   //screen width
#define height 700   //screen height
#define fishNumber 10
#define maxThreds 1024
#define maxBlocks 100
#define M_PI 3.14159265358979323846
#define MaxSpeed 10
constexpr float CohesionScale = 0.01f;
constexpr float AlignmentScale = 0.1f;
constexpr float SeparationScale = 0.1f;


struct Shoal {
    float* position_x;
    float* position_y;
    float* velocity_x;
    float* velocity_y;
    int h = 30;
    int w = 5;
    int minDistance = 20;
    int viewRange = 100;
};

struct Grid {
    int* cellsId;
    int* firstFishInCell;
    int* lastFishInCell;
    int gridNumber_Vertical;
    int gridNumber_Horyzontal;
    int gridWidth;
    int gridHeight;
};

struct Point {
    double x;
    double y;
};


int x = 0;
float t = 0.0f;
float* device;   //pointer to memory on the device (GPU VRAM)
GLuint buffer;   //buffer
Shoal shoal;
Grid grid;
float background;
float fishColor;

static unsigned int CompileShader(unsigned int type, const std::string &source)
{
    unsigned int id = glCreateShader(type);
    const char* src = source.c_str();
    glShaderSource(id, 1, &src, nullptr);
    glCompileShader(id);

    int result;
    glGetShaderiv(id, GL_COMPILE_STATUS, &result);
    if (result == GL_FALSE)
    {
        int length;
        glGetShaderiv(id, GL_INFO_LOG_LENGTH, &length);
        char* message = (char*)alloca(length * sizeof(char));
        glGetShaderInfoLog(id, length, &length, message);
        std::cout << "Failed to compile shader!" << std::endl;
        std::cout << message << std::endl;
        glDeleteShader(id);
        return 0;
    }

    return id;
    
}

static unsigned int CreateShader(const std::string& vertexShader, const std::string& fragmentShader)
{
    unsigned int program = glCreateProgram();
    unsigned int vs = CompileShader(GL_VERTEX_SHADER, vertexShader);
    unsigned int fs = CompileShader(GL_FRAGMENT_SHADER, fragmentShader);
    glAttachShader(program, vs);
    glAttachShader(program, fs);
    glLinkProgram(program);
    glValidateProgram(program);
    glDeleteShader(vs);
    glDeleteShader(fs);
    return program;
}

__device__ Point MakePoint(float x, float y)
{
    Point p;
    p.x = x;
    p.y = y;
    return p;
}

__device__ double Distance(Point p1, Point p2)
{
    return pow(p1.x - p2.x, (double)2) + pow(p1.y - p2.y, (double)2);
}

__device__ float Direction(Point vel)
{
    float direction = asin(vel.y / sqrt(Distance(MakePoint(0, 0), vel)));
    
    if (vel.x < 0)
    {
        if (vel.y >= 0)
        {
            float alfa = M_PI / 2 - direction;
            direction += 2 * alfa;
        }
        else
        {
            float alfa = M_PI / 2 + direction;
            direction -= 2 * alfa;
        }
    }
    return direction;
}

__device__ void FishToCordinates(Point fishPosition, float direction, int h, int w, Point* p1, Point* p2, Point* p3)
{
    float hX = cos(direction) * h / 2;
    float hY = sin(direction) * h / 2;

    p1->x = fishPosition.x + hX;
    p1->y = fishPosition.y + hY;

    float fishWingX = -sin(direction) * w;
    float fishWingY = cos(direction) * w;

    p2->x = fishPosition.x - hX + fishWingX;
    p2->y = fishPosition.y - hY + fishWingY;

    p3->x = fishPosition.x - hX - fishWingX;
    p3->y = fishPosition.y - hY - fishWingY;
}


__device__ int FishsCellId(Point fish, int gridWidth, int gridHeight, int gridNumber_Horyzontal)
{
    int gridX = fish.x / gridWidth;
    int gridY = fish.y / gridHeight;
    return gridY * gridNumber_Horyzontal + gridX;
}



__global__ void CalculateShoal(Shoal shoal, Grid grid, float* output)
{
    unsigned int fishId = blockIdx.x * blockDim.x + threadIdx.x;
    if (fishId >= fishNumber)
        return;

    float viewRange = shoal.viewRange * shoal.viewRange;
    float minDistance = shoal.minDistance * shoal.minDistance;


    Point fish = MakePoint(shoal.position_x[fishId], shoal.position_y[fishId]);
    Point centerOfMass = MakePoint(0, 0);
    Point separation = MakePoint(0, 0);
    Point avrVelocity = MakePoint(0, 0);
    int neighboursCountCenterOfMass = 0;
    int neighboursCountVelocity = 0;
    
    for (int x = fish.x - viewRange; x <= fish.x + viewRange; x += grid.gridWidth)
    {
        for (int y = fish.y - viewRange; y <= fish.y + viewRange; x += grid.gridHeight)
        {
            int cellId = FishsCellId(MakePoint(x, y), grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);

            int start = grid.firstFishInCell[cellId];
            int end = grid.lastFishInCell[cellId];
            if (start == -1)
                continue;

            for (int neighbourFishId = start; neighbourFishId <= end; neighbourFishId++)
            {

                if (neighbourFishId == fishId)
                    continue;

                Point fishsFriend = MakePoint(shoal.position_x[neighbourFishId], shoal.position_y[neighbourFishId]);
                double distance = Distance(fish, fishsFriend);


                if (viewRange > distance)
                {
                    neighboursCountCenterOfMass++;
                    centerOfMass.x += fishsFriend.x;
                    centerOfMass.y += fishsFriend.y;

                }

                if (viewRange > distance)
                {
                    neighboursCountVelocity++;
                    avrVelocity.x += shoal.velocity_x[neighbourFishId];
                    avrVelocity.y += shoal.velocity_x[neighbourFishId];
                }

                if (minDistance > distance)
                {
                    separation.x -= fishsFriend.x - fish.x;
                    separation.y -= fishsFriend.y - fish.y;
                }
            }
        }
    }
    Point newVelocity = MakePoint(shoal.velocity_x[fishId], shoal.velocity_y[fishId]);
    if (neighboursCountCenterOfMass > 0)
    {
        centerOfMass.x /= neighboursCountCenterOfMass;
        centerOfMass.y /= neighboursCountCenterOfMass;

        newVelocity.x += (centerOfMass.x - fish.x) * CohesionScale;
        newVelocity.y += (centerOfMass.y - fish.y) * CohesionScale;
    }
    
    separation.x *= SeparationScale;
    separation.y *= SeparationScale;

    newVelocity.x += separation.x;
    newVelocity.y += separation.y;

    if (neighboursCountVelocity != 0) {
        avrVelocity.x /= neighboursCountVelocity;
        avrVelocity.y /= neighboursCountVelocity;
        newVelocity.x += avrVelocity.x * AlignmentScale;
        newVelocity.y += avrVelocity.y * AlignmentScale;
    }

    if (Distance(MakePoint(0, 0), newVelocity) > MaxSpeed * MaxSpeed)
    {
        double calculateSpeed = sqrt(Distance(MakePoint(0, 0), newVelocity));
        newVelocity.x *= MaxSpeed / calculateSpeed;
        newVelocity.y *= MaxSpeed / calculateSpeed;
    }

    fish.x += newVelocity.x;
    fish.y += newVelocity.y;


    // zapisz nowe ustawienie rybki
    if (fish.x < 0)
        shoal.position_x[fishId] = width;
    else if (fish.x > width)
        shoal.position_x[fishId] = 0;
    else 
        shoal.position_x[fishId] = fish.x;

    if (fish.y < 0)
        shoal.position_y[fishId] = height;
    else if (fish.y > height)
        shoal.position_y[fishId] = 0;
    else
        shoal.position_y[fishId] = fish.y;

    shoal.velocity_x[fishId] = newVelocity.x;
    shoal.velocity_y[fishId] = newVelocity.y;

    // zapisuje id gripu
    
    grid.cellsId[fishId] = FishsCellId(fish, grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);

    // zapisuje kordynaty rybki
    Point p1, p2, p3;
    FishToCordinates(fish, Direction(newVelocity), shoal.h, shoal.w, &p1, &p2, &p3);
    int start = fishId * 6;
    output[start] = p1.x;
    output[start + 1] = p1.y;
    output[start + 2] = p2.x;
    output[start + 3] = p2.y;
    output[start + 4] = p3.x;
    output[start + 5] = p3.y;
}



__global__ void InitStartPosition(Shoal shoal, int fisheGrideWidth, int fishGrideHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= fishNumber)
        return;

    int distance = shoal.minDistance;
    int startPoint_x = (width - distance*fisheGrideWidth) / 2;
    int startPoimt_y = (height - distance*fishGrideHeight) / 2;
    int fishGridY = x / fisheGrideWidth;
    int fishGridX = x - fishGridY * fisheGrideWidth;
    shoal.velocity_x[x] = MaxSpeed * cos(x * 2* M_PI / fishNumber);
    shoal.velocity_y[x] = MaxSpeed * sin(x * 2 * M_PI / fishNumber);
    shoal.position_x[x] = startPoint_x + fishGridX * distance;
    shoal.position_y[x] = startPoimt_y + fishGridY * distance;
}

__global__ void ResetGridStartEnd(int* start, int* end)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= fishNumber)
        return;
    start[x] = -1;
    end[x] = -1;
}

__global__ void CalculateStartEnd(int* start, int* end, int* gridId)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= fishNumber)
        return;

    int curentGridId = gridId[x];

    if (x == 0 || gridId[x - 1] != curentGridId)
        start[curentGridId] = x;

    if (x == fishNumber - 1 || gridId[x + 1] != curentGridId)
        end[curentGridId] = x;

}




void time(int x)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(17, time, 0);
        t += 0.0166f;
    }
}

void CalculateNeededThreads(int* threads, int* blocks)
{
    if (fishNumber < maxThreds)
    {
        *threads = fishNumber;
        *blocks = 1;
    }
    else
    {
        *threads = maxThreds;
        *blocks = ceil(fishNumber / maxThreds);
        if (*blocks > maxBlocks)
            exit(-1);
    }
}

void LunchCuda()
{
    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.position_x);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.position_y);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.velocity_x);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.velocity_y);
    thrust::sort(thrust::device, grid.cellsId, grid.cellsId + fishNumber);

    ResetGridStartEnd<< <blocks, threads >> > (grid.firstFishInCell, grid.lastFishInCell);
    hipDeviceSynchronize();
    CalculateStartEnd << <blocks, threads >> > (grid.firstFishInCell, grid.lastFishInCell, grid.cellsId);
    hipDeviceSynchronize();
    CalculateShoal << <blocks, threads >> > (shoal, grid, device);
    hipDeviceSynchronize();

}
// Display callback function
void display() {
    // Clear the window
    cudaGLMapBufferObject((void**)&device, buffer);   //maps the buffer object into the address space of CUDA
    glClear(GL_COLOR_BUFFER_BIT);

    LunchCuda();
    hipDeviceSynchronize();

    cudaGLUnmapBufferObject(buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glDrawArrays(GL_TRIANGLES, 0, 3*fishNumber);
    glutSwapBuffers();
    
    x++;
    if (x > width)
        x = 0;
}

void InitCuda()
{
    grid.gridHeight = (shoal.viewRange + 1)/ 2;
    grid.gridWidth = grid.gridHeight;
    hipMalloc(&device, fishNumber * 6 * sizeof(float));   //allocate memory on the GPU VRAM
    hipMalloc(&shoal.position_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.position_y, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_y, fishNumber * sizeof(float));
    hipMalloc(&grid.cellsId, fishNumber * sizeof(int));
    hipMalloc(&grid.firstFishInCell, fishNumber * sizeof(int));
    hipMalloc(&grid.lastFishInCell, fishNumber * sizeof(int));
    grid.gridNumber_Horyzontal = ceil((double)width / (double)grid.gridWidth);
    grid.gridNumber_Vertical = ceil((double)height / (double)grid.gridHeight);
    int fisheGrideWidth = ceil(sqrt((fishNumber * width) / height));
    int fishGrideHeight = ceil(fisheGrideWidth * height / width);

    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks);
    InitStartPosition << <blocks, threads >> > (shoal, fisheGrideWidth, fishGrideHeight);
}

void Init()
{
    glClearColor(0.0, 0.0, 0.0, 0.0);
    glMatrixMode(GL_PROJECTION);
    gluOrtho2D(0.0, width, 0.0, height);
    glutDisplayFunc(display);
    //glutReshapeFunc(Reshape);
    time(0);
    glewInit();
    glGenBuffers(1, &buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    unsigned int size = fishNumber * 6 * sizeof(float); // ilość wektorów
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 2, 0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    std::string vertexShader =
        "#version 330 core\n"
        "layout(location = e) in vec4 position; \n"
        "\n"
        "void main()\n"
        "{\n"
        "gl_Position = position; \n"
        "}\n";
    std::string fragmentShader =
        "#version 330 core\n"
        "\n"
        "layout(location = 0) out vec4 color; \n"
        "\n"
        "void main()\n"
        "{\n"
        " color = vec4(0.1, 1.0, 0.1, 1.0); \n"
        "}\n";

    unsigned int shader = CreateShader(vertexShader, fragmentShader);
    glUseProgram(shader);

    InitCuda();
    cudaGLRegisterBufferObject(buffer);   //register the buffer object for access by CUDA
}

void FreeShoalOfFish()
{
    hipFree(device);
    hipFree(grid.lastFishInCell);
    hipFree(grid.firstFishInCell);
    hipFree(grid.cellsId);
    hipFree(shoal.velocity_x);
    hipFree(shoal.velocity_y);
    hipFree(shoal.velocity_x);
    hipFree(shoal.velocity_y);
}

int main(int argc, char** argv) {
    // Initialize GLUT
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);   //display mode
    glutInitWindowSize(width, height);
    glutCreateWindow("ShoalOfFish"); // Create the window
    Init();
    // Enter the main loop
    glutMainLoop();
    FreeShoalOfFish();
    return 0;
}