#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <cmath>
#include <string>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#define width 1280   //screen width
#define height 700   //screen height
#define fishNumber 100
#define maxThreds 1024
#define maxBlocks 100
#define M_PI 3.14159265358979323846
#define MaxSpeed 3
constexpr float CohesionScale = 0.01f;
constexpr float AlignmentScale = 0.1f;
constexpr float SeparationScale = 0.1f;


struct Shoal {
    float* position_x;
    float* position_y;
    float* velocity_x;
    float* velocity_y;
    int h = 12;
    int w = 3;
    int minDistance = 20;
    int viewRange = 100;
};

struct Grid {
    int* cellsId;
    int* firstFishInCell;
    int* lastFishInCell;
    int gridNumber_Vertical;
    int gridNumber_Horyzontal;
    int gridWidth;
    int gridHeight;
};

struct Point {
    double x;
    double y;
};


float t = 0.0f;
float* device;   //pointer to memory on the device (GPU VRAM)
GLuint buffer;   //buffer
Shoal shoal;
Grid grid;
float background;
float fishColor;

static unsigned int CompileShader(unsigned int type, const std::string &source)
{
    unsigned int id = glCreateShader(type);
    const char* src = source.c_str();
    glShaderSource(id, 1, &src, nullptr);
    glCompileShader(id);

    int result;
    glGetShaderiv(id, GL_COMPILE_STATUS, &result);
    if (result == GL_FALSE)
    {
        int length;
        glGetShaderiv(id, GL_INFO_LOG_LENGTH, &length);
        char* message = (char*)alloca(length * sizeof(char));
        glGetShaderInfoLog(id, length, &length, message);
        std::cout << "Failed to compile shader!" << std::endl;
        std::cout << message << std::endl;
        glDeleteShader(id);
        return 0;
    }

    return id;
    
}

static unsigned int CreateShader(const std::string& vertexShader, const std::string& fragmentShader)
{
    unsigned int program = glCreateProgram();
    unsigned int vs = CompileShader(GL_VERTEX_SHADER, vertexShader);
    unsigned int fs = CompileShader(GL_FRAGMENT_SHADER, fragmentShader);
    glAttachShader(program, vs);
    glAttachShader(program, fs);
    glLinkProgram(program);
    glValidateProgram(program);
    glDeleteShader(vs);
    glDeleteShader(fs);
    return program;
}

__device__ Point MakePoint(float x, float y)
{
    Point p;
    p.x = x;
    p.y = y;
    return p;
}

__device__ double PowDistance(Point p1, Point p2)
{
    return pow(p1.x - p2.x, (double)2) + pow(p1.y - p2.y, (double)2);
}

__device__ float Direction(Point vel)
{
    float direction = asin(vel.y / sqrt(PowDistance(MakePoint(0, 0), vel)));
    
    if (vel.x < 0)
    {
        if (vel.y >= 0)
        {
            float alfa = M_PI / 2 - direction;
            direction += 2 * alfa;
        }
        else
        {
            float alfa = M_PI / 2 + direction;
            direction -= 2 * alfa;
        }
    }
    return direction;
}

__device__ void FishToCordinates(Point fishPosition, float direction, int h, int w, Point* p1, Point* p2, Point* p3)
{
    float hX = cos(direction) * h / 2;
    float hY = sin(direction) * h / 2;

    p1->x = fishPosition.x + hX;
    p1->y = fishPosition.y + hY;

    float fishWingX = -sin(direction) * w;
    float fishWingY = cos(direction) * w;

    p2->x = fishPosition.x - hX + fishWingX;
    p2->y = fishPosition.y - hY + fishWingY;

    p3->x = fishPosition.x - hX - fishWingX;
    p3->y = fishPosition.y - hY - fishWingY;
}


__device__ int FishsCellId(Point fish, int gridWidth, int gridHeight, int gridNumber_Horyzontal)
{
    int gridX = fish.x / gridWidth;
    int gridY = fish.y / gridHeight;
    return gridY * gridNumber_Horyzontal + gridX;
}

__global__ void CategorizeFishToCells(Shoal shoal, Grid grid)
{
    unsigned int fishId = blockIdx.x * blockDim.x + threadIdx.x;
    if (fishId >= fishNumber)
        return;

    Point fish = MakePoint(shoal.position_x[fishId], shoal.position_y[fishId]);

    grid.cellsId[fishId] = FishsCellId(fish, grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);
}


__global__ void CalculateShoal(Shoal shoal, Grid grid, float* output)
{
    unsigned int fishId = blockIdx.x * blockDim.x + threadIdx.x;
    if (fishId >= fishNumber)
        return;

    int powViewRange = shoal.viewRange * shoal.viewRange;
    int powMinDistance = shoal.minDistance * shoal.minDistance;


    Point fish = MakePoint(shoal.position_x[fishId], shoal.position_y[fishId]);
    Point newVelocity = MakePoint(shoal.velocity_x[fishId], shoal.velocity_y[fishId]);
    Point centerOfMass = MakePoint(0, 0);
    Point separation = MakePoint(0, 0);
    Point avrVelocity = MakePoint(0, 0);
    int neighboursCountCenterOfMass = 0;
    int neighboursCountVelocity = 0;

    int leftDownCornerCell = FishsCellId(MakePoint(fish.x - shoal.viewRange, fish.y - shoal.viewRange),
        grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);
    int rightDownCornerCell = FishsCellId(MakePoint(fish.x + shoal.viewRange, fish.y - shoal.viewRange),
        grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);
    int leftUpCornerCell = FishsCellId(MakePoint(fish.x - shoal.viewRange, fish.y + shoal.viewRange),
        grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);

    int horyzontalCells = rightDownCornerCell - leftDownCornerCell;
    int verticalCells = leftUpCornerCell - leftDownCornerCell;



    for (int x = 0; x <= horyzontalCells; x++)
    {
        if (x < 0) { continue; }
        for (int y = 0; y <= verticalCells; y++)
        {
            if (y < 0) { continue; }
            //int cellId = FishsCellId(MakePoint(x, y), grid.gridWidth, grid.gridHeight, grid.gridNumber_Horyzontal);
            int cellId = y * grid.gridWidth + leftDownCornerCell + x;

            if (cellId > grid.gridWidth * grid.gridHeight)
            {
                continue;
            }
            int start = grid.firstFishInCell[cellId];
            int end = grid.lastFishInCell[cellId];
            if (start == -1)
                continue;

            for (int neighbourFishId = start; neighbourFishId <= end; neighbourFishId++)
            {

                if (neighbourFishId == fishId)
                    continue;

                Point fishsFriend = MakePoint(shoal.position_x[neighbourFishId], shoal.position_y[neighbourFishId]);
                double powDistance = PowDistance(fish, fishsFriend);


                if (powViewRange > powDistance)
                {
                    neighboursCountCenterOfMass++;
                    centerOfMass.x += fishsFriend.x;
                    centerOfMass.y += fishsFriend.y;

                }

                if (powViewRange > powDistance)
                {
                    neighboursCountVelocity++;
                    avrVelocity.x += shoal.velocity_x[neighbourFishId];
                    avrVelocity.y += shoal.velocity_x[neighbourFishId];
                }

                if (powMinDistance > powDistance)
                {
                    separation.x -= fishsFriend.x - fish.x;
                    separation.y -= fishsFriend.y - fish.y;
                }
            }
        }
    }

    
    if (neighboursCountCenterOfMass > 0)
    {
        centerOfMass.x /= neighboursCountCenterOfMass;
        centerOfMass.y /= neighboursCountCenterOfMass;

        newVelocity.x += (centerOfMass.x - fish.x) * CohesionScale;
        newVelocity.y += (centerOfMass.y - fish.y) * CohesionScale;
    }
    
    separation.x *= SeparationScale;
    separation.y *= SeparationScale;

    newVelocity.x += separation.x;
    newVelocity.y += separation.y;

    if (neighboursCountVelocity != 0) {
        avrVelocity.x /= neighboursCountVelocity;
        avrVelocity.y /= neighboursCountVelocity;
        newVelocity.x += avrVelocity.x * AlignmentScale;
        newVelocity.y += avrVelocity.y * AlignmentScale;
    }

    if (PowDistance(MakePoint(0, 0), newVelocity) > MaxSpeed * MaxSpeed)
    {
        double calculateSpeed = sqrt(PowDistance(MakePoint(0, 0), newVelocity));
        newVelocity.x *= MaxSpeed / calculateSpeed;
        newVelocity.y *= MaxSpeed / calculateSpeed;
    }

    fish.x += newVelocity.x;
    fish.y += newVelocity.y;


    // zapisz nowe ustawienie rybki
    if (fish.x < 0)
        shoal.position_x[fishId] = width;
    else if (fish.x > width)
        shoal.position_x[fishId] = 0;
    else 
        shoal.position_x[fishId] = fish.x;

    if (fish.y < 0)
        shoal.position_y[fishId] = height;
    else if (fish.y > height)
        shoal.position_y[fishId] = 0;
    else
        shoal.position_y[fishId] = fish.y;

    shoal.velocity_x[fishId] = newVelocity.x;
    shoal.velocity_y[fishId] = newVelocity.y;

    

    // zapisuje kordynaty rybki
    Point p1, p2, p3;
    FishToCordinates(fish, Direction(newVelocity), shoal.h, shoal.w, &p1, &p2, &p3);
    int start = fishId * 6;
    output[start] = p1.x;
    output[start + 1] = p1.y;
    output[start + 2] = p2.x;
    output[start + 3] = p2.y;
    output[start + 4] = p3.x;
    output[start + 5] = p3.y;

}



__global__ void InitStartPosition(Shoal shoal, int fisheGrideWidth, int fishGrideHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= fishNumber)
        return;

    int distance = shoal.minDistance;
    int startPoint_x = (width - distance*fisheGrideWidth) / 2;
    int startPoimt_y = (height - distance*fishGrideHeight) / 2;
    int fishGridY = x / fisheGrideWidth;
    int fishGridX = x - fishGridY * fisheGrideWidth;
    shoal.velocity_x[x] = MaxSpeed * cos(x * 2* M_PI / fishNumber);
    shoal.velocity_y[x] = MaxSpeed * sin(x * 2 * M_PI / fishNumber);
    shoal.position_x[x] = startPoint_x + fishGridX * distance;
    shoal.position_y[x] = startPoimt_y + fishGridY * distance;
}

__global__ void ResetGridStartEnd(int* start, int* end)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= fishNumber)
        return;
    start[x] = -1;
    end[x] = -1;
}

__global__ void CalculateStartEnd(int* start, int* end, int* gridId)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= fishNumber)
        return;

    int curentGridId = gridId[x];

    if (x == 0 || gridId[x - 1] != curentGridId)
        start[curentGridId] = x;

    if (x == fishNumber - 1 || gridId[x + 1] != curentGridId)
        end[curentGridId] = x;
}




void time(int x)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(100, time, 0);
        t += 0.0166f;
    }
}

void CalculateNeededThreads(int* threads, int* blocks, int neededThreads)
{
    if (neededThreads < maxThreds)
    {
        *threads = neededThreads;
        *blocks = 1;
    }
    else
    {
        *threads = maxThreds;
        *blocks = (int)ceil((double)neededThreads / maxThreds);
        if (*blocks > maxBlocks)
            exit(-1);
    }
}

void LunchCuda()
{
    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks, fishNumber);
    CategorizeFishToCells << <blocks, threads >> > (shoal, grid);
    hipDeviceSynchronize();

    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.position_x);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber , shoal.position_y);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.velocity_x);
    thrust::sort_by_key(thrust::device, grid.cellsId, grid.cellsId + fishNumber, shoal.velocity_y);
    thrust::sort(thrust::device, grid.cellsId, grid.cellsId + fishNumber);

    CalculateNeededThreads(&threads, &blocks, grid.gridWidth * grid.gridHeight);
    ResetGridStartEnd<< <blocks, threads >> > (grid.firstFishInCell, grid.lastFishInCell);
    hipDeviceSynchronize();

    CalculateNeededThreads(&threads, &blocks, fishNumber);
    CalculateStartEnd << <blocks, threads >> > (grid.firstFishInCell, grid.lastFishInCell, grid.cellsId);
    hipDeviceSynchronize();

    CalculateNeededThreads(&threads, &blocks, fishNumber);
    CalculateShoal << <blocks, threads >> > (shoal, grid, device);
    hipDeviceSynchronize();
}
// Display callback function
void display() {
    // Clear the window
    cudaGLMapBufferObject((void**)&device, buffer);   //maps the buffer object into the address space of CUDA
    glClear(GL_COLOR_BUFFER_BIT);

    LunchCuda();

    cudaGLUnmapBufferObject(buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glDrawArrays(GL_TRIANGLES, 0, 3*fishNumber);
    glutSwapBuffers();
    
}

void InitCuda()
{
    grid.gridHeight = (shoal.viewRange + 1)/ 2;
    grid.gridWidth = grid.gridHeight;
    hipMalloc(&device, fishNumber * 6 * sizeof(float));   //allocate memory on the GPU VRAM
    hipMalloc(&shoal.position_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.position_y, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_x, fishNumber * sizeof(float));
    hipMalloc(&shoal.velocity_y, fishNumber * sizeof(float));
    hipMalloc(&grid.cellsId, fishNumber * sizeof(int));
    hipMalloc(&grid.firstFishInCell, grid.gridHeight * grid.gridWidth * sizeof(int));
    hipMalloc(&grid.lastFishInCell, grid.gridHeight * grid.gridWidth * sizeof(int));
    grid.gridNumber_Horyzontal = (int)ceil((double)width / (double)grid.gridWidth);
    grid.gridNumber_Vertical = (int)ceil((double)height / (double)grid.gridHeight);
    

    int blocks, threads;
    CalculateNeededThreads(&threads, &blocks, fishNumber);
    int rantagleOfFishWidth = (int)ceil(sqrt((fishNumber * width) / height));
    int rantagleOfFishHeight = (int)ceil(rantagleOfFishWidth * height / width);
    InitStartPosition << <blocks, threads >> > (shoal, rantagleOfFishWidth, rantagleOfFishHeight);
}

void Init()
{
    glClearColor(0.0, 0.0, 0.0, 0.0);
    glMatrixMode(GL_PROJECTION);
    gluOrtho2D(0.0, width, 0.0, height);
    glutDisplayFunc(display);
    //glutReshapeFunc(Reshape);
    time(0);
    glewInit();
    glGenBuffers(1, &buffer);
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    unsigned int size = fishNumber * 6 * sizeof(float); // ilość wektorów
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 2, 0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    std::string vertexShader =
        "#version 330 core\n"
        "layout(location = e) in vec4 position; \n"
        "\n"
        "void main()\n"
        "{\n"
        "gl_Position = position; \n"
        "}\n";
    std::string fragmentShader =
        "#version 330 core\n"
        "\n"
        "layout(location = 0) out vec4 color; \n"
        "\n"
        "void main()\n"
        "{\n"
        " color = vec4(0.1, 1.0, 0.1, 1.0); \n"
        "}\n";

    unsigned int shader = CreateShader(vertexShader, fragmentShader);
    glUseProgram(shader);

    InitCuda();
    cudaGLRegisterBufferObject(buffer);   //register the buffer object for access by CUDA
}

void FreeShoalOfFish()
{
    hipFree(device);
    hipFree(grid.lastFishInCell);
    hipFree(grid.firstFishInCell);
    hipFree(grid.cellsId);
    hipFree(shoal.velocity_x);
    hipFree(shoal.velocity_y);
    hipFree(shoal.velocity_x);
    hipFree(shoal.velocity_y);
}

int main(int argc, char** argv) {
    // Initialize GLUT
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);   //display mode
    glutInitWindowSize(width, height);
    glutCreateWindow("ShoalOfFish"); // Create the window
    Init();
    // Enter the main loop
    glutMainLoop();
    FreeShoalOfFish();
    return 0;
}